#include <armadillo>
#include <cstdlib>
#include <cublasXt.h>
#include <omp.h>
#include <stdexcept>

#include "linalg.hpp"


namespace gpublas
{
  static const int blockdim = 2048;
  static const float cpu_ratio = 0.0f;
  
  
  
  namespace err
  {
    inline std::string get_cublas_error_msg(hipblasStatus_t check)
    {
      if (check == HIPBLAS_STATUS_SUCCESS)
        return "";
      else if (check == HIPBLAS_STATUS_NOT_INITIALIZED)
        return "cuBLAS not initialized";
      else if (check == HIPBLAS_STATUS_ALLOC_FAILED)
        return "internal cuBLAS memory allocation failed";
      else if (check == HIPBLAS_STATUS_INVALID_VALUE)
        return "unsupported parameter";
      else if (check == HIPBLAS_STATUS_ARCH_MISMATCH)
        return "function requires feature missing from device architecture";
      else if (check == HIPBLAS_STATUS_MAPPING_ERROR)
        return "access to GPU memory space failed";
      else if (check == HIPBLAS_STATUS_EXECUTION_FAILED)
        return "GPU program failed to execute";
      else if (check == HIPBLAS_STATUS_INTERNAL_ERROR)
        return "internal cuBLAS operation failed";
      else if (check == HIPBLAS_STATUS_NOT_SUPPORTED)
        return "requested functionality is not supported";
      else if (check == HIPBLAS_STATUS_UNKNOWN)
        return "error with cuBLAS license check";
      else
        return "unknown cuBLAS error occurred";
    }
    
    inline void check_ret(hipblasStatus_t check, std::string op)
    {
      if (check != HIPBLAS_STATUS_SUCCESS)
      {
        std::string msg = "cuBLAS " + op + "() failed with error: " + get_cublas_error_msg(check);
        throw std::runtime_error(msg);
      }
    }
  }
  
  
  
  arma::fmat xtx_gemm(const float alpha, const arma::fmat &x, const int num_gpus)
  {
    const int m = x.n_rows;
    const int n = x.n_cols;
    arma::fmat ret(n, n);
    
    
    hipblasStatus_t st;
    
    cublasXtHandle_t h;
    st = cublasXtCreate(&h);
    err::check_ret(st, "xtgemm");
    
    st = cublasXtSetBlockDim(h, blockdim);
    err::check_ret(st, "xtgemm");
    
    int dev_id[num_gpus];
    for (int i=0; i<num_gpus; i++)
      dev_id[i] = i;
    st = cublasXtDeviceSelect(h, num_gpus, dev_id);
    err::check_ret(st, "xtgemm");
    
    st = cublasXtSetCpuRatio(h, CUBLASXT_GEMM, CUBLASXT_FLOAT, cpu_ratio);
    err::check_ret(st, "xtgemm");
    
    const float beta = 0.0f;
    st = cublasXtSgemm(h, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, m, &alpha, 
      x.memptr(), m, x.memptr(), m, &beta, ret.memptr(), n);
    err::check_ret(st, "xtgemm");
    
    st = cublasXtDestroy(h);
    err::check_ret(st, "xtgemm");
    
    return ret;
  }
  
  
  
  arma::fmat xtx_syrk(const float alpha, const arma::fmat &x, const int num_gpus)
  {
    const int m = x.n_rows;
    const int n = x.n_cols;
    arma::fmat ret(n, n);
    
    
    hipblasStatus_t st;
    
    cublasXtHandle_t h;
    st = cublasXtCreate(&h);
    err::check_ret(st, "xtsyrk");
    
    st = cublasXtSetBlockDim(h, blockdim);
    err::check_ret(st, "xtsyrk");
    
    int dev_id[num_gpus];
    for (int i=0; i<num_gpus; i++)
      dev_id[i] = i;
    st = cublasXtDeviceSelect(h, num_gpus, dev_id);
    err::check_ret(st, "xtsyrk");
    
    st = cublasXtSetCpuRatio(h, CUBLASXT_GEMM, CUBLASXT_FLOAT, cpu_ratio);
    err::check_ret(st, "xtsyrk");
    
    const float beta = 0.0f;
    st = cublasXtSsyrk(h, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, n, m, &alpha, 
      x.memptr(), m, &beta, ret.memptr(), n);
    err::check_ret(st, "xtsyrk");
    
    st = cublasXtDestroy(h);
    err::check_ret(st, "xtsyrk");
    
    cpublas::symmetrize(ret);
    
    return ret;
  }
}
